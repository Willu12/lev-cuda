#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <cstdio>

#define ALPHABET_SIZE 95

__global__ void create_x_matrix(int* x_matrix,const char* word,const char* alphabet, int size) {
    int tid = threadIdx.x;
    if(tid >= ALPHABET_SIZE) return;
    
    for(int j =0; j<size; j++) {

        if(j == 0) {
            x_matrix[tid] = 0;
            continue;
        }
        
        printf("%d\n",size);
        int index = tid * size + j;
        printf("index = %d, (size + 1) * ALPHABET_SIZE = %d\n",index,(size +1) * ALPHABET_SIZE);
        x_matrix[index] = word[j - 1] == alphabet[tid] ? j : x_matrix[index - 1];
    }
}

__device__ int calculate_d_value(int* d_matrix, char* word1, char* word2, int* x_matrix, int current_index, int length) {
    
    int i = current_index /length;
    int j = current_index % length;

    if(i == 0) return j;
    if(j == 0) return i;

    if(word1[j - 1] == word2[i - 1]) return d_matrix[current_index - 1 - length];

    // wiemy że litera m = word2[i -1] znajduje się na miejscu w alfabecie będącym jej wartością ASCII 
    int l = word2[i - 1] - 32;

    if(x_matrix[l + j * ALPHABET_SIZE] == 0) {
        return 1 + min(d_matrix[current_index - length],
        min(d_matrix[current_index - length - 1],i + j -1));
    }

    return 1 + min(
        d_matrix[current_index - length],
        min(d_matrix[current_index - length - 1],
        d_matrix[(i - 1) * length + x_matrix[l * ALPHABET_SIZE + j] - 1] + (j - 1 - x_matrix[l * ALPHABET_SIZE + j]))
    );
}

__global__ void create_d_matrix(int* d_matrix, char* word1, char* word2, int* x_matrix, int size, int length) {

    //okej wywolujemy tyle wątkow ile liter ma slowo 1
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // wywalmy tidy ktore są za duże
    if(tid > length -1) return;
    for(int i =0; i<size + 1; i++) {

        d_matrix[tid + length * i] = calculate_d_value(d_matrix, word1, word2,x_matrix,tid + length * i,length);
        __syncthreads();
    }
}



int* create_X_matrix(char* word, int len) {
    
    // we assume that word consists only of ascii characters
    int * x_matrix;
    char* alphabet_device;
    const std::string ALPHABET = std::string(" !\"#$%&\'()*+,-./0123456789:;<=>?@ABCDEFGHIJKLMNOPQRSTUVWXYZ[\\]^_`abcdefghijklmnopqrstuvwxyz{|}~");

    hipError_t cudaStatus;

    cudaStatus = hipMalloc(&x_matrix, sizeof(int) * (1 + len) * ALPHABET.size());
    hipMalloc(&alphabet_device, sizeof(char) * ALPHABET.size());
    
    cudaStatus = hipMemcpy(alphabet_device, ALPHABET.data(), ALPHABET.size() * sizeof(char), hipMemcpyHostToDevice);
    
    create_x_matrix<<<1,256>>>(x_matrix,word,alphabet_device,len);


    return x_matrix;
}

int* create_D_matrix(char* word1,char* word2, int len1, int len2,int *x_matrix) {
    int* d_matrix;

    hipMalloc(&d_matrix, sizeof(int) * (len1 + 1) * (len2 + 1));
    
    const int threads_per_blocks = 512;

    const int blocks = len1  / threads_per_blocks + !!(len1 % threads_per_blocks);

    create_d_matrix<<<blocks,threads_per_blocks>>>(d_matrix,word1,word2,x_matrix,len2,len1);
    

    return d_matrix;
}