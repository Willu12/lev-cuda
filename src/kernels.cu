#include "hip/hip_runtime.h"
#include "kernels.cuh"

#define ALPHABET_SIZE 95

__global__ void create_x_matrix(int* x_matrix, char* word, char* alphabet, int size) {
    int tid = threadIdx.x;

    for(int j =0; j<size; j++) {

        if(tid == 0 || j == 0) {
            x_matrix[tid] = 0;
            continue;
        }

        int index = tid * size + j;

        x_matrix[index] = word[j - 1] == alphabet[tid] ? j : x_matrix[index - 1];
    }
}

__device__ int calculate_d_value(int* d_matrix, char* word1, char* word2, int* x_matrix, int current_index, int length) {
    
    int i = current_index /length;
    int j = current_index % length;

    if(i == 0) return j;
    if(j == 0) return i;

    if(word1[j - 1] == word2[i - 1]) return d_matrix[current_index - 1 - length];

    // wiemy że litera m = word2[i -1] znajduje się na miejscu w alfabecie będącym jej wartością ASCII 
    int l = word2[i - 1] - 32;

    if(x_matrix[l + j * ALPHABET_SIZE] == 0) {
        return 1 + min(d_matrix[current_index - length],
        min(d_matrix[current_index - length - 1],i + j -1));
    }

    return 1 + min(
        d_matrix[current_index - length],
        min(d_matrix[current_index - length - 1],
        d_matrix[(i - 1) * length + x_matrix[l * ALPHABET_SIZE + j] - 1] + (j - 1 - x_matrix[l * ALPHABET_SIZE + j]))
    );
}

__global__ void create_d_matrix(int* d_matrix, char* word1, char* word2, int* x_matrix, int size, int length) {

    //okej wywolujemy tyle wątkow ile liter ma slowo 1
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for(int i =0; i<size + 1; i++) {

        d_matrix[tid + length * i] = calculate_d_value(d_matrix, word1, word2,x_matrix,tid + length * i,length);
        __syncthreads();
    }
}



int* create_X_matrix(char* word, int len) {
    
    // we assume that word consists only of ascii characters
    int * x_matrix;
    char* alphabet_device;

    hipMalloc(&x_matrix, sizeof(int) * (1 + len) * ALPHABET.size());
    hipMalloc(&alphabet_device, sizeof(char) * ALPHABET.size());

    hipMemcpy(alphabet_device, ALPHABET.data(), ALPHABET.size(), hipMemcpyHostToDevice);

    create_x_matrix<<<1,256>>>(x_matrix,word,alphabet_device,len);


    return x_matrix;
}

int* create_D_matrix(char* word1,char* word2, int len1, int len2,int *x_matrix) {
    int* d_matrix;

    hipMalloc(&d_matrix, sizeof(int) * (len1 + 1) * (len2 + 1));
    
    

    return d_matrix;
}